#include "hip/hip_runtime.h"
/***

Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the "Software"),
to deal in the Software without restriction, including without limitation
the rights to use, copy, modify, merge, publish, distribute, sublicense,
and/or sell copies of the Software, and to permit persons to whom the
Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
IN THE SOFTWARE.

***/

#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <gmp.h>
#include "cgbn/cgbn.h"
#include "../utility/support.h"
#include "powm_odd_export.h"

// For this example, there are quite a few template parameters that are used to generate the actual code.
// In order to simplify passing many parameters, we use the same approach as the CGBN library, which is to
// create a container class with static constants and then pass the class.

// The CGBN context uses the following three parameters:
//   TBP             - threads per block (zero means to use the blockDim.x)
//   MAX_ROTATION    - must be small power of 2, imperically, 4 works well
//   SHM_LIMIT       - number of bytes of dynamic shared memory available to the kernel
//   CONSTANT_TIME   - require constant time algorithms (currently, constant time algorithms are not available)

// Locally it will also be helpful to have several parameters:
//   TPI             - threads per instance
//   BITS            - number of bits per instance
//   WINDOW_BITS     - number of bits to use for the windowed exponentiation

template<uint32_t tpi, uint32_t bits, uint32_t window_bits>
class powm_params_t {
  public:
  // parameters used by the CGBN context
  static const uint32_t TPB=0;                     // get TPB from blockDim.x  
  static const uint32_t MAX_ROTATION=4;            // good default value
  static const uint32_t SHM_LIMIT=0;               // no shared mem available
  static const bool     CONSTANT_TIME=false;       // constant time implementations aren't available yet
  
  // parameters used locally in the application
  static const uint32_t TPI=tpi;                   // threads per instance
  static const uint32_t BITS=bits;                 // instance size
  static const uint32_t WINDOW_BITS=window_bits;   // window size
};

template<class params>
class powm_odd_t {
  public:
  static const uint32_t window_bits=params::WINDOW_BITS;  // used a lot, give it an instance variable

  // It might be possible to switch to a SOA structure within the instance_t struct
  // Currently, I believe removing this struct completely would make things worse
  // The main advantage of the current interleaved AOS input structure is that it allows making the 
  // input memory longer by concatenating byte arrays that represent valid inputs
  // I also need to run benchmarks on an x16 pcie link to make sure we're making the correct pcie bandwidth tradeoff
  // Results shouldn't belong in the instance struct. They should get allocated and written separately, so as to not
  // have to download and uploaded more than is necessary. x and pow should only be uploaded, and results should only
  // be downloaded.
  typedef struct {
    cgbn_mem_t<params::BITS> x;
    cgbn_mem_t<params::BITS> power;
  } input_t;
  
  typedef cgbn_context_t<params::TPI, params>   context_t;
  typedef cgbn_env_t<context_t, params::BITS>   env_t;
  typedef typename env_t::cgbn_t                bn_t;
  typedef typename env_t::cgbn_local_t          bn_local_t;

  context_t _context;
  env_t     _env;
  int32_t   _instance;

  __device__ __forceinline__ powm_odd_t(cgbn_monitor_t monitor, cgbn_error_report_t *report, int32_t instance) : _context(monitor, report, (uint32_t)instance), _env(_context), _instance(instance) {
  }

  __device__ __forceinline__ void fixed_window_powm_odd(bn_t &result, const bn_t &x, const bn_t &power, const bn_t &modulus) {
    bn_t       t;
    bn_local_t window[1<<window_bits];
    int32_t    index, position, offset;
    uint32_t   np0;

    // conmpute x^power mod modulus, using the fixed window algorithm
    // requires:  x<modulus,  modulus is odd

    // compute x^0 (in Montgomery space, this is just 2^BITS - modulus)
    cgbn_negate(_env, t, modulus);
    cgbn_store(_env, window+0, t);
    
    // convert x into Montgomery space, store into window table
    np0=cgbn_bn2mont(_env, result, x, modulus);
    cgbn_store(_env, window+1, result);
    cgbn_set(_env, t, result);
    
    // compute x^2, x^3, ... x^(2^window_bits-1), store into window table
    #pragma nounroll
    for(index=2;index<(1<<window_bits);index++) {
      cgbn_mont_mul(_env, result, result, t, modulus, np0);
      cgbn_store(_env, window+index, result);
    }

    // find leading high bit
    position=params::BITS - cgbn_clz(_env, power);

    // break the exponent into chunks, each window_bits in length
    // load the most significant non-zero exponent chunk
    offset=position % window_bits;
    if(offset==0)
      position=position-window_bits;
    else
      position=position-offset;
    index=cgbn_extract_bits_ui32(_env, power, position, window_bits);
    cgbn_load(_env, result, window+index);
    
    // process the remaining exponent chunks
    while(position>0) {
      // square the result window_bits times
      #pragma nounroll
      for(int sqr_count=0;sqr_count<window_bits;sqr_count++)
        cgbn_mont_sqr(_env, result, result, modulus, np0);
      
      // multiply by next exponent chunk
      position=position-window_bits;
      index=cgbn_extract_bits_ui32(_env, power, position, window_bits);
      cgbn_load(_env, t, window+index);
      cgbn_mont_mul(_env, result, result, t, modulus, np0);
    }
    
    // we've processed the exponent now, convert back to normal space
    cgbn_mont2bn(_env, result, result, modulus, np0);
  }
  
  __device__ __forceinline__ void sliding_window_powm_odd(bn_t &result, const bn_t &x, const bn_t &power, const bn_t &modulus) {
    bn_t         t, starts;
    int32_t      index, position, leading;
    uint32_t     mont_inv;
    bn_local_t   odd_powers[1<<window_bits-1];

    // compute x^power mod modulus, using Constant Length Non-Zero windows (CLNZ).
    // requires:  x<modulus,  modulus is odd
        
    // find the leading one in the power
    leading=params::BITS-1-cgbn_clz(_env, power);
    if(leading>=0) {
      // convert x into Montgomery space, store in the odd powers table
      mont_inv=cgbn_bn2mont(_env, result, x, modulus);
      
      // compute t=x^2 mod modulus
      cgbn_mont_sqr(_env, t, result, modulus, mont_inv);
      
      // compute odd powers window table: x^1, x^3, x^5, ...
      cgbn_store(_env, odd_powers, result);
      #pragma nounroll
      for(index=1;index<(1<<window_bits-1);index++) {
        cgbn_mont_mul(_env, result, result, t, modulus, mont_inv);
        cgbn_store(_env, odd_powers+index, result);
      }
  
      // starts contains an array of bits indicating the start of a window
      cgbn_set_ui32(_env, starts, 0);
  
      // organize p as a sequence of odd window indexes
      position=0;
      while(true) {
        if(cgbn_extract_bits_ui32(_env, power, position, 1)==0)
          position++;
        else {
          cgbn_insert_bits_ui32(_env, starts, starts, position, 1, 1);
          if(position+window_bits>leading)
            break;
          position=position+window_bits;
        }
      }
  
      // load first window.  Note, since the window index must be odd, we have to
      // divide it by two before indexing the window table.  Instead, we just don't
      // load the index LSB from power
      index=cgbn_extract_bits_ui32(_env, power, position+1, window_bits-1);
      cgbn_load(_env, result, odd_powers+index);
      position--;
      
      // Process remaining windows 
      while(position>=0) {
        cgbn_mont_sqr(_env, result, result, modulus, mont_inv);
        if(cgbn_extract_bits_ui32(_env, starts, position, 1)==1) {
          // found a window, load the index
          index=cgbn_extract_bits_ui32(_env, power, position+1, window_bits-1);
          cgbn_load(_env, t, odd_powers+index);
          cgbn_mont_mul(_env, result, result, t, modulus, mont_inv);
        }
        position--;
      }
      
      // convert result from Montgomery space
      cgbn_mont2bn(_env, result, result, modulus, mont_inv);
    }
    else {
      // p=0, thus x^p mod modulus=1
      cgbn_set_ui32(_env, result, 1);
    }
  }
};

// kernel implementation using cgbn
// 
// Unfortunately, the kernel must be separate from the powm_odd_t class
// kernel_powm_odd<params><<<(instance_count+IPB-1)/IPB, TPB>>>(report, gpuInputs, gpuResults, instance_count);
template<class params>
__global__ void kernel_powm_odd(cgbn_error_report_t *report, typename powm_odd_t<params>::input_t *inputs, cgbn_mem_t<params::BITS> *modulus, cgbn_mem_t<params::BITS> *outputs, uint32_t count) {
  int32_t instance;

  // decode an instance number from the blockIdx and threadIdx
  instance=(blockIdx.x*blockDim.x + threadIdx.x)/params::TPI;
  if(instance>=count)
    return;

  powm_odd_t<params>                 po(cgbn_report_monitor, report, instance);
  typename powm_odd_t<params>::bn_t  r, x, p, m;
  
  // the loads and stores can go in the class, but it seems more natural to have them
  // here and to pass in and out bignums
  cgbn_load(po._env, x, &(inputs[instance].x));
  cgbn_load(po._env, p, &(inputs[instance].power));
  cgbn_load(po._env, m, modulus);
  
  // this can be either fixed_window_powm_odd or sliding_window_powm_odd.
  // when TPI<32, fixed window runs much faster because it is less divergent, so we use it here
  // po.fixed_window_powm_odd(r, x, p, m);
  //   OR
  po.sliding_window_powm_odd(r, x, p, m);
  
  cgbn_store(po._env, &(outputs[instance]), r);
}

// Result of upload_powm
template<class params>
struct powm_upload_results_t {
  // Number of items: instance_count
  typename powm_odd_t<params>::input_t *gpuInputs;
  cgbn_mem_t<params::BITS> *gpuResults;
  // Number of items: 1
  cgbn_mem_t<params::BITS> *gpuModulus;

  uint32_t instance_count;
  cgbn_error_report_t *report;
  // Wait on this event for uploading to finish
  hipEvent_t event;
};

// Check error before proceeding
// Does async memcpy set the error?
// Clean up struct if error is present
// Uploads memory from host to device, asynchronously
// Returns a struct that will contain the necessary parameters to the run function
// FIXME This should be part of the powm class, right?
// Returns error
// Puts resulting valid structure (except in error cases) in last parameter
template<class params>
const char* upload_powm(const void* modulus, const void *inputs, const uint32_t instance_count, powm_upload_results_t<params>* result) {
  typedef typename powm_odd_t<params>::input_t input_t;
  
  // Set instance count; it's re-used when the kernel gets run later
  result->instance_count = instance_count;
  // Initialize some fields to null
  // If an error occurs, non-null GPU buffers should be cleaned up by the caller
  result->gpuInputs = NULL;
  result->gpuResults = NULL;
  result->gpuModulus = NULL;
  result->report = NULL;
  result->event = NULL;
  
  CUDA_CHECK_RETURN(hipSetDevice(0));
  // 1 modulus per kernel invocation
  const size_t modulusSize = sizeof(cgbn_mem_t<params::BITS>);
  // instance_count results per kernel invocation
  const size_t resultsSize = sizeof(cgbn_mem_t<params::BITS>)*instance_count;
  // instance_count inputs per kernel invocation
  const size_t inputsSize = sizeof(input_t)*instance_count;

  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK_RETURN(cgbn_error_report_alloc(&(result->report)));

  // About hipEventDisableTiming, not getting timing information from the
  // event is supposed to give better performance when waiting on it
  CUDA_CHECK_RETURN(hipEventCreateWithFlags(&result->event, hipEventDisableTiming));

  // TODO these should belong with a stream and should be allocated and freed
  //  much less often
  CUDA_CHECK_RETURN(hipMalloc((void **)&(result->gpuInputs), inputsSize));
  CUDA_CHECK_RETURN(hipMalloc((void **)&(result->gpuResults), resultsSize));
  CUDA_CHECK_RETURN(hipMalloc((void **)&(result->gpuModulus), modulusSize));

  // TODO make which stream these copies happen on controllable from bindings
  //  Currently, the stream argument is just null
  CUDA_CHECK_RETURN(hipMemcpyAsync((void *)result->gpuInputs, inputs, inputsSize, hipMemcpyHostToDevice));

  // Currently, we're copying to the modulus before each kernel launch
  CUDA_CHECK_RETURN(hipMemcpyAsync((void *)result->gpuModulus, modulus, modulusSize, hipMemcpyHostToDevice));

  // Run should wait on this event for kernel launch
  // The event should include any memcpys that haven't completed yet
  CUDA_CHECK_RETURN(hipEventRecord(result->event));

  return NULL;
}

// Run powm kernel
// Blocks until kernel execution finishes, then copies results from device to host
// To call this, you should have prepared a kernel launch with upload_powm
// and waited for the returned struct to be populated
// The method will only work properly with a valid (i.e. non-error) 
// powm_upload_results_t
// The results will be placed in the passed results pointer after the kernel run
template<class params>
const char* run_powm(const powm_upload_results_t<params> *upload, void *results) {
  // TODO Wait on upload event to finish before running kernel
  //  Can't be done until we switch to async uploads
  typedef typename powm_odd_t<params>::input_t input_t;

  const int32_t              TPB=(params::TPB==0) ? 128 : params::TPB;    // default threads per block to 128
  const int32_t              TPI=params::TPI, IPB=TPB/TPI;                // IPB is instances per block

  // We have instance_count results, each is a certain number of bits wide
  const size_t resultsSize = sizeof(cgbn_mem_t<params::BITS>)*upload->instance_count;

  CUDA_CHECK_RETURN(hipSetDevice(0));
  CUDA_CHECK_RETURN(hipEventSynchronize(upload->event));

  // launch kernel with blocks=ceil(instance_count/IPB) and threads=TPB
  kernel_powm_odd<params><<<(upload->instance_count+IPB-1)/IPB, TPB>>>(
    upload->report, 
    upload->gpuInputs, 
    upload->gpuModulus, 
    upload->gpuResults, 
    upload->instance_count);

  // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
  // Note: This should probably only happen in debug builds, as the error 
  // report might not be necessary in normal usage
  CUDA_CHECK_RETURN(hipDeviceSynchronize());
  CGBN_CHECK_RETURN(upload->report);

  // The kernel ran successfully, so we get the results off the GPU
  CUDA_CHECK_RETURN(hipMemcpy(results, upload->gpuResults, resultsSize, hipMemcpyDeviceToHost));

  // We don't need these GPU buffers anymore, as the kernel has run
  // TODO Re-use resources rather than creating or destroying them every time
  CUDA_CHECK_RETURN(hipFree((void*)upload->gpuInputs));
  CUDA_CHECK_RETURN(hipFree((void*)upload->gpuResults));
  CUDA_CHECK_RETURN(hipFree((void*)upload->gpuModulus));
  CUDA_CHECK_RETURN(cgbn_error_report_free(upload->report));
  CUDA_CHECK_RETURN(hipEventDestroy(upload->event));
  return NULL;
}

typedef powm_params_t<32, 4096, 5> params4096;

template<class params>
inline return_data* powm_export(const powm_upload_results_t<params> *upload) {
  // Run kernel
  return_data *rd = (return_data*)malloc(sizeof(*rd));
  auto result_mem = malloc(sizeof(cgbn_mem_t<params::BITS>) * upload->instance_count);
  rd->error = run_powm<params>(upload, result_mem);
  rd->result = result_mem;
  return rd;
}

template<class params>
inline return_data* upload_export(const void *prime, const void *instances, const uint32_t instance_count) {
  // Upload data
  return_data *rd = (return_data*)malloc(sizeof(*rd));
  auto up = (powm_upload_results_t<params>*)malloc(sizeof(powm_upload_results_t<params>));
  rd->error = upload_powm<params>(prime, instances, instance_count, up);
  if (rd->error == NULL) {
    // Normal case
    rd->result = up;
  } else {
    // Error case
    rd->result = NULL;
    // Attempt to free non-null buffers: if there was an error, the whole 
    // upload shouldn't be valid, so they're no longer useful
    if (up->report != NULL) {
      cgbn_error_report_free(up->report);
    }
    if (up->gpuInputs != NULL) {
      hipFree(up->gpuInputs);
    }
    if (up->gpuModulus != NULL) {
      hipFree(up->gpuModulus);
    }
    if (up->gpuResults != NULL) {
      hipFree(up->gpuResults);
    }
    free(up);
  }
  return rd;
}

// Call this when starting the program to allocate resources
// Returns pointer to instance and error
// bitLength should be a multiple of 32
template<class params>
inline const char* createStreamManager(numStreams, bitLength, capacity, streamManager *streams) {
  streams->numStreams = numStreams;
  streams->streams = (typeof(streams->streams))malloc(sizeof(*streams->streams)*numStreams);
  for (int i = 0; i < numStreams; i++) {
    CUDA_CHECK_RETURN(hipStreamCreate(&streams->streams[i].stream));
    CUDA_CHECK_RETURN(hipMalloc((void **)&(streams->streams[i].gpuInputs), inputsSize));
    CUDA_CHECK_RETURN(hipMalloc((void **)&(streams->streams[i].gpuResults), resultsSize));
    CUDA_CHECK_RETURN(hipMalloc((void **)&(streams->streams[i].gpuModulus), modulusSize));
    streams->streams[i].capacity = capacity;
    streams->streams[i].length = 0;
    CUDA_CHECK_RETURN(cgbn_error_report_alloc(&streams->streams[i].report));
    // These events are created without timing data because timing data
    // hinders performance, and we don't need the timing data.
    CUDA_CHECK_RETURN(hipEventCreateWithFlags(&(streams->streams[i].hostToDevice), hipEventDisableTiming));
    CUDA_CHECK_RETURN(hipEventCreateWithFlags(&(streams->streams[i].exec), hipEventDisableTiming));
    CUDA_CHECK_RETURN(hipEventCreateWithFlags(&(streams->streams[i].deviceToHost), hipEventDisableTiming));
  }
  // Upload result should be a pointer to stream manager and an index to the
  // specific stream's data.
}

template<class params>
inline const char* destroyStreamManager(streamManager *streams) {
  // After program execution, the stream manager is no longer needed, and
  // should get cleaned up.
  for (int i = 0; i < streams->numStreams; i++) {
    CUDA_CHECK_RETURN(hipFree(streams->streams[i].gpuModulus));
    CUDA_CHECK_RETURN(hipFree(streams->streams[i].gpuModulus));
    CUDA_CHECK_RETURN(hipFree(streams->streams[i].gpuModulus));
    CUDA_CHECK_RETURN(cgbn_error_report_free(&streams->streams[i].report));
    CUDA_CHECK_RETURN(hipEventDestroy(streams->streams[i].hostToDevice));
    CUDA_CHECK_RETURN(hipEventDestroy(streams->streams[i].exec));
    CUDA_CHECK_RETURN(hipEventDestroy(streams->streams[i].deviceToHost));
    free(streams->streams[i]);
  }
  free(streams);
}

// Stream object and associated data for a stream
template<class params>
struct streamData {
  // This CUDA stream; when performing operations, set the current stream to
  // this one
  hipStream_t stream;

  // Number of items is number of instances
  typename powm_odd_t<params>::input_t *gpuInputs;
  cgbn_mem_t<params::BITS> *gpuResults;
  // Number of items is 1
  cgbn_mem_t<params::BITS> *gpuModulus;

  // Number of items that can be held in the buffers associated with this stream
  uint32_t capacity;
  // Number of items to be processed with this part of the stream
  uint32_t length;

  // Check for CGBN errors after kernel finishes using this
  cgbn_error_report_t *report;
  // Synchronize this event to wait for host to device transfer before kernel execution
  hipEvent_t hostToDevice;
  // Synchronize this event to wait for kernel execution to finish before device to host transfer
  hipEvent_t exec;
  // Synchronize this event to wait for downloading to finish before using results
  hipEvent_t deviceToHost;
};

template<class params>
struct streamManager {
  uint32_t numStreams;
  streamData<params> *streams;
};

// All the methods used in cgo should have extern "C" linkage to avoid
// implementation-specific name mangling
// This makes them more straightforward to load from the shared object
extern "C" {
  // Upload data for a powm kernel run for 4K bits
  return_data* upload_powm_4096(const void *prime, const void *instances, const uint32_t instance_count) {
    return upload_export<params4096>(prime, instances, instance_count);
  }
  
  // Run powm for 4K bits
  return_data* run_powm_4096(const void *upload_result) {
    return powm_export<params4096>((powm_upload_results_t<params4096>*)upload_result);
  }


  // Call this after execution has completed to deallocate resources
  // Returns error
  const char* destroyPowm(void *instance) {
  }

  // Call this after execution has completed to write out profile information to the disk
  const char* stopProfiling() {
    CUDA_CHECK_RETURN(hipProfilerStop());
    return NULL;
  }

  const char* startProfiling() {
    CUDA_CHECK_RETURN(hipProfilerStart());
    return NULL;
  }

  const char* resetDevice() {
    CUDA_CHECK_RETURN(hipDeviceReset());
    return NULL;
  }
}

